// Copyright (c) 2025, Jingze Shi and Tri Dao.
// Splitting the different head dimensions to different files to speed up compilation.
// This file is auto-generated. See "generate_kernels.py"#include "namespace_config.h"
#include "flash_fwd_launch_template.h"

namespace FLASH_NAMESPACE {

template<>
void run_mha_fwd_<cutlass::bfloat16_t, 96, true, false, true>(Flash_fwd_params &params, hipStream_t stream) {
    run_mha_fwd_hdim96<cutlass::bfloat16_t, true, false, true>(params, stream);
}

} // namespace FLASH_NAMESPACE