#include "hip/hip_runtime.h"
// DEM kernels that does some wildcard stuff, such as modifying the system as per user instruction
#include <DEMHelperKernels.cuh>
#include <DEM/Defines.h>
_kernelIncludes_;

// Mass properties are below, if jitified mass properties are in use
_massDefs_;
_moiDefs_;

__global__ void applyFamilyChanges(deme::DEMSimParams* simParams, deme::DEMDataDT* granData, size_t nOwnerBodies) {
    deme::bodyID_t myOwner = blockIdx.x * blockDim.x + threadIdx.x;
    if (myOwner < nOwnerBodies) {
        // The user may make references to owner positions, velocities, accelerations and simulation time
        double3 pos;
        float3 vel, acc;
        float mass;
        deme::family_t family_code = granData->familyID[myOwner];
        // Get my mass info from either jitified arrays or global memory
        // Outputs myMass
        // Use an input named exactly `myOwner' which is the id of this owner
        {
            float myMass;
            _massAcqStrat_;
            mass = myMass;
        }
        voxelIDToPosition<double, deme::voxelID_t, deme::subVoxelPos_t>(
            pos.x, pos.y, pos.z, granData->voxelID[myOwner], granData->locX[myOwner], granData->locY[myOwner],
            granData->locZ[myOwner], _nvXp2_, _nvYp2_, _voxelSize_, _l_);
        pos.x += simParams->LBFX;
        pos.y += simParams->LBFY;
        pos.z += simParams->LBFZ;

        vel.x = granData->vX[myOwner];
        vel.y = granData->vY[myOwner];
        vel.z = granData->vZ[myOwner];
        acc.x = granData->aX[myOwner];
        acc.y = granData->aY[myOwner];
        acc.z = granData->aZ[myOwner];

        // Standardize names...
        double X = pos.x;
        double Y = pos.y;
        double Z = pos.z;
        float vX = vel.x;
        float vY = vel.y;
        float vZ = vel.z;
        float accX = acc.x;
        float accY = acc.y;
        float accZ = acc.z;

        float ts = simParams->h;
        float time = simParams->timeElapsed;

        // Carry out user's instructions
        { _familyChangeRules_; }
    }
}
