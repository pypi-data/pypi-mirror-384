#include "hip/hip_runtime.h"
// DEM integration related custom kernels
#include <DEMHelperKernels.cuh>
#include <DEM/Defines.h>
_kernelIncludes_;

// Apply presecibed velocity and report whether the `true' physics should be skipped, rather than added on top of
// that
template <typename T1, typename T2, typename T3, typename T4>
inline __device__ void applyPrescribedVel(bool& LinVelXPrescribed,
                                          bool& LinVelYPrescribed,
                                          bool& LinVelZPrescribed,
                                          bool& RotVelXPrescribed,
                                          bool& RotVelYPrescribed,
                                          bool& RotVelZPrescribed,
                                          T1& vX,
                                          T1& vY,
                                          T1& vZ,
                                          T2& omgBarX,
                                          T2& omgBarY,
                                          T2& omgBarZ,
                                          T3 X,
                                          T3 Y,
                                          T3 Z,
                                          T4 oriQw,
                                          T4 oriQx,
                                          T4 oriQy,
                                          T4 oriQz,
                                          deme::bodyID_t ownerID,
                                          const deme::family_t& family,
                                          const float& t) {
    switch (family) {
        _velPrescriptionStrategy_;
        default:
            // Default can just do nothing
            return;
    }
}

// Apply presecibed location and report whether the `true' physics should be skipped, rather than added on top of that
template <typename T1, typename T2, typename T3, typename T4>
inline __device__ void applyPrescribedPos(bool& LinXPrescribed,
                                          bool& LinYPrescribed,
                                          bool& LinZPrescribed,
                                          bool& RotPrescribed,
                                          T1& X,
                                          T1& Y,
                                          T1& Z,
                                          T2& oriQw,
                                          T2& oriQx,
                                          T2& oriQy,
                                          T2& oriQz,
                                          T3 vX,
                                          T3 vY,
                                          T3 vZ,
                                          T4 omgBarX,
                                          T4 omgBarY,
                                          T4 omgBarZ,
                                          deme::bodyID_t ownerID,
                                          const deme::family_t& family,
                                          const float& t) {
    switch (family) {
        _posPrescriptionStrategy_;
        default:
            // Default can just do nothing
            return;
    }
}

// Apply extra accelerations for family numbers
template <typename T1, typename T2, typename T3, typename T4, typename T5, typename T6>
inline __device__ void applyAddedAcceleration(T1& accX,
                                              T1& accY,
                                              T1& accZ,
                                              T2& angAccX,
                                              T2& angAccY,
                                              T2& angAccZ,
                                              T3 X,
                                              T3 Y,
                                              T3 Z,
                                              T4 oriQw,
                                              T4 oriQx,
                                              T4 oriQy,
                                              T4 oriQz,
                                              T5 vX,
                                              T5 vY,
                                              T5 vZ,
                                              T6 omgBarX,
                                              T6 omgBarY,
                                              T6 omgBarZ,
                                              deme::bodyID_t ownerID,
                                              const deme::family_t& family,
                                              const float& t) {
    switch (family) {
        _accPrescriptionStrategy_;
        default:
            return;
    }
}

inline __device__ void integrateVelPos(deme::bodyID_t ownerID,
                                       deme::DEMSimParams* simParams,
                                       deme::DEMDataDT* granData,
                                       float3& v,
                                       float3& omgBar,
                                       float h,
                                       float t) {
    // Acquisition phase...
    deme::family_t family_code = granData->familyID[ownerID];
    bool LinVelXPrescribed = false, LinVelYPrescribed = false, LinVelZPrescribed = false, RotVelXPrescribed = false,
         RotVelYPrescribed = false, RotVelZPrescribed = false;
    bool LinXPrescribed = false, LinYPrescribed = false, LinZPrescribed = false, RotPrescribed = false;
    double X, Y, Z;
    // Keep tab of the old... we'll need that
    float3 old_v = make_float3(granData->vX[ownerID], granData->vY[ownerID], granData->vZ[ownerID]);
    float3 old_omgBar = make_float3(granData->omgBarX[ownerID], granData->omgBarY[ownerID], granData->omgBarZ[ownerID]);

    {
        // Now XYZ gets the old position. We can write them directly back, then it is equivalent to being LinPrescribed.
        voxelIDToPosition<double, deme::voxelID_t, deme::subVoxelPos_t>(
            X, Y, Z, granData->voxelID[ownerID], granData->locX[ownerID], granData->locY[ownerID],
            granData->locZ[ownerID], _nvXp2_, _nvYp2_, _voxelSize_, _l_);
        // Do this and we get the `true' pos... Needed for prescription
        X += (double)simParams->LBFX;
        Y += (double)simParams->LBFY;
        Z += (double)simParams->LBFZ;

        // The user may directly change v and omgBar info in global memory in applyPrescribedVel (XYZ and oriQ in this
        // call are read-only)
        applyPrescribedVel(LinVelXPrescribed, LinVelYPrescribed, LinVelZPrescribed, RotVelXPrescribed,
                           RotVelYPrescribed, RotVelZPrescribed, granData->vX[ownerID], granData->vY[ownerID],
                           granData->vZ[ownerID], granData->omgBarX[ownerID], granData->omgBarY[ownerID],
                           granData->omgBarZ[ownerID], X, Y, Z, granData->oriQw[ownerID], granData->oriQx[ownerID],
                           granData->oriQy[ownerID], granData->oriQz[ownerID], ownerID, family_code, (float)t);
        // The user may directly change oriQ info (vX and omgBar in this call are read-only)
        applyPrescribedPos(LinXPrescribed, LinYPrescribed, LinZPrescribed, RotPrescribed, X, Y, Z,
                           granData->oriQw[ownerID], granData->oriQx[ownerID], granData->oriQy[ownerID],
                           granData->oriQz[ownerID], granData->vX[ownerID], granData->vY[ownerID],
                           granData->vZ[ownerID], granData->omgBarX[ownerID], granData->omgBarY[ownerID],
                           granData->omgBarZ[ownerID], ownerID, family_code, (float)t);
    }

    // Operation phase...

    {
        // User's addition of accelerations won't affect acc arrays in global memory; that is, if the user query the
        // contact acceleration, still they don't get the part they applied in this acc prescription
        float3 v_update = make_float3(0, 0, 0), omgBar_update = make_float3(0, 0, 0);
        float3 extra_acc = make_float3(0, 0, 0), extra_angAcc = make_float3(0, 0, 0);
        applyAddedAcceleration(extra_acc.x, extra_acc.y, extra_acc.z, extra_angAcc.x, extra_angAcc.y, extra_angAcc.z, X,
                               Y, Z, granData->oriQw[ownerID], granData->oriQx[ownerID], granData->oriQy[ownerID],
                               granData->oriQz[ownerID], granData->vX[ownerID], granData->vY[ownerID],
                               granData->vZ[ownerID], granData->omgBarX[ownerID], granData->omgBarY[ownerID],
                               granData->omgBarZ[ownerID], ownerID, family_code, (float)t);

        if (!LinVelXPrescribed) {
            v_update.x = (granData->aX[ownerID] + extra_acc.x + simParams->Gx) * h;
            granData->vX[ownerID] += v_update.x;
        } else {
            old_v.x = granData->vX[ownerID];
        }
        if (!LinVelYPrescribed) {
            v_update.y = (granData->aY[ownerID] + extra_acc.y + simParams->Gy) * h;
            granData->vY[ownerID] += v_update.y;
        } else {
            old_v.y = granData->vY[ownerID];
        }
        if (!LinVelZPrescribed) {
            v_update.z = (granData->aZ[ownerID] + extra_acc.z + simParams->Gz) * h;
            granData->vZ[ownerID] += v_update.z;
        } else {
            old_v.z = granData->vZ[ownerID];
        }

        if (!RotVelXPrescribed) {
            omgBar_update.x = (granData->alphaX[ownerID] + extra_angAcc.x) * h;
            granData->omgBarX[ownerID] += omgBar_update.x;
        } else {
            old_omgBar.x = granData->omgBarX[ownerID];
        }
        if (!RotVelYPrescribed) {
            omgBar_update.y = (granData->alphaY[ownerID] + extra_angAcc.y) * h;
            granData->omgBarY[ownerID] += omgBar_update.y;
        } else {
            old_omgBar.y = granData->omgBarY[ownerID];
        }
        if (!RotVelZPrescribed) {
            omgBar_update.z = (granData->alphaZ[ownerID] + extra_angAcc.z) * h;
            granData->omgBarZ[ownerID] += omgBar_update.z;
        } else {
            old_omgBar.z = granData->omgBarZ[ownerID];
        }

        // We need to set v and omgBar, and they will be used in position/quaternion update
        _integrationVelocityPassOnStrategy_;
    }

    // With v and omgBar. update pos now...
    {
        if (!LinXPrescribed) {
            // Impllicitly, pos integration strategy is here
            X += (double)v.x * h;
        }
        if (!LinYPrescribed) {
            Y += (double)v.y * h;
        }
        if (!LinZPrescribed) {
            Z += (double)v.z * h;
        }
        // Undo the influence of LBF...
        X -= (double)simParams->LBFX;
        Y -= (double)simParams->LBFY;
        Z -= (double)simParams->LBFZ;
        positionToVoxelID<deme::voxelID_t, deme::subVoxelPos_t, double>(
            granData->voxelID[ownerID], granData->locX[ownerID], granData->locY[ownerID], granData->locZ[ownerID], X, Y,
            Z, _nvXp2_, _nvYp2_, _voxelSize_, _l_);

        if (!RotPrescribed) {
            // Then integrate the quaternion
            // 1st Taylor series multiplier. First use it to record delta rotation...
            // Refer to
            // https://stackoverflow.com/questions/24197182/efficient-quaternion-angular-velocity/24201879#24201879
            const float3 ha = 0.5 * h * omgBar;
            float4 oriQ = make_float4(ha.x, ha.y, ha.z, 1.0);  // xyzw
            // Note: Yes it is Quat * deltaRot, not the other way around. Then store result in oriQ.
            HamiltonProduct(oriQ.w, oriQ.x, oriQ.y, oriQ.z, granData->oriQw[ownerID], granData->oriQx[ownerID],
                            granData->oriQy[ownerID], granData->oriQz[ownerID], oriQ.w, oriQ.x, oriQ.y, oriQ.z);
            // Normalizing it is essential. Note even if you use an exp map to update quaternion, you still need to
            // normalize.
            oriQ /= length(oriQ);
            granData->oriQw[ownerID] = oriQ.w;
            granData->oriQx[ownerID] = oriQ.x;
            granData->oriQy[ownerID] = oriQ.y;
            granData->oriQz[ownerID] = oriQ.z;
        }
    }
}

// inline __device__ void locateNewVoxel(deme::voxelID_t& voxel, int64_t& locX_tmp, int64_t& locY_tmp, int64_t&
// locZ_tmp) {
//     deme::voxelID_t voxelX;
//     deme::voxelID_t voxelY;
//     deme::voxelID_t voxelZ;
//     IDChopper<deme::voxelID_t, deme::voxelID_t>(voxelX, voxelY, voxelZ, voxel, _nvXp2_, _nvYp2_);

//     // MAX_SUBVOXEL is int64 and large enough to handle VOXEL_RES_POWER2 == 16 or 32
//     voxelX += div_floor<int64_t, int64_t>(locX_tmp, deme::MAX_SUBVOXEL);
//     voxelY += div_floor<int64_t, int64_t>(locY_tmp, deme::MAX_SUBVOXEL);
//     voxelZ += div_floor<int64_t, int64_t>(locZ_tmp, deme::MAX_SUBVOXEL);
//     locX_tmp = mod_floor<int64_t, int64_t>(locX_tmp, deme::MAX_SUBVOXEL);
//     locY_tmp = mod_floor<int64_t, int64_t>(locY_tmp, deme::MAX_SUBVOXEL);
//     locZ_tmp = mod_floor<int64_t, int64_t>(locZ_tmp, deme::MAX_SUBVOXEL);

//     IDPacker<deme::voxelID_t, deme::voxelID_t>(voxel, voxelX, voxelY, voxelZ, _nvXp2_, _nvYp2_);
// }

__global__ void integrateOwners(deme::DEMSimParams* simParams, deme::DEMDataDT* granData) {
    deme::bodyID_t ownerID = blockIdx.x * blockDim.x + threadIdx.x;
    if (ownerID < simParams->nOwnerBodies) {
        // These 2 quantities mean the velocity and ang vel used for updating position/quaternion for this step.
        // Depending on the integration scheme in use, they can be different.
        float3 v, omgBar;
        integrateVelPos(ownerID, simParams, granData, v, omgBar, (float)simParams->h, (float)simParams->timeElapsed);
    }
}
