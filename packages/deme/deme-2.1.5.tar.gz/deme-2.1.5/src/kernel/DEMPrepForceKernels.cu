#include "hip/hip_runtime.h"
// DEM force computation related custom kernels
#include <DEM/Defines.h>
#include <DEMHelperKernels.cuh>
_kernelIncludes_;

inline __device__ void cleanUpContactForces(size_t thisContact,
                                            deme::DEMSimParams* simParams,
                                            deme::DEMDataDT* granData) {
    const float3 zeros = make_float3(0, 0, 0);
    granData->contactForces[thisContact] = zeros;
    granData->contactTorque_convToForce[thisContact] = zeros;
}

inline __device__ void cleanUpAcc(size_t thisClump, deme::DEMSimParams* simParams, deme::DEMDataDT* granData) {
    // If should not clear acc arrays, then just mark it to be clear in the next ts
    if (granData->accSpecified[thisClump]) {
        granData->accSpecified[thisClump] = 0;
    } else {
        granData->aX[thisClump] = 0;
        granData->aY[thisClump] = 0;
        granData->aZ[thisClump] = 0;
    }
    if (granData->angAccSpecified[thisClump]) {
        granData->angAccSpecified[thisClump] = 0;
    } else {
        granData->alphaX[thisClump] = 0;
        granData->alphaY[thisClump] = 0;
        granData->alphaZ[thisClump] = 0;
    }
}

__global__ void prepareAccArrays(deme::DEMSimParams* simParams, deme::DEMDataDT* granData) {
    size_t myID = blockIdx.x * blockDim.x + threadIdx.x;
    if (myID < simParams->nOwnerBodies) {
        cleanUpAcc(myID, simParams, granData);
    }
}

__global__ void prepareForceArrays(deme::DEMSimParams* simParams, deme::DEMDataDT* granData, size_t nContactPairs) {
    size_t myID = blockIdx.x * blockDim.x + threadIdx.x;
    if (myID < nContactPairs) {
        cleanUpContactForces(myID, simParams, granData);
    }
}

__global__ void rearrangeContactWildcards(deme::DEMDataDT* granData,
                                          float* newWildcards,
                                          deme::notStupidBool_t* sentry,
                                          unsigned int nWildcards,
                                          size_t nContactPairs) {
    size_t myID = blockIdx.x * blockDim.x + threadIdx.x;
    if (myID < nContactPairs) {
        deme::contactPairs_t map_from = granData->contactMapping[myID];
        if (map_from == deme::NULL_MAPPING_PARTNER) {
            // If it is a NULL ID then kT says this contact is new. Initialize all wildcard arrays.
            for (size_t i = 0; i < nWildcards; i++) {
                newWildcards[nContactPairs * i + myID] = 0;
            }
        } else {
            // Not a new contact, need to map it from somewhere in the old history array
            for (size_t i = 0; i < nWildcards; i++) {
                newWildcards[nContactPairs * i + myID] = granData->contactWildcards[i][map_from];
            }
            // This sentry trys to make sure that all `alive' contacts got mapped to some place
            sentry[map_from] = 0;
        }
    }
}

__global__ void markAliveContacts(float* wildcard, deme::notStupidBool_t* sentry, size_t nContactPairs) {
    size_t myID = blockIdx.x * blockDim.x + threadIdx.x;
    if (myID < nContactPairs) {
        float myEntry = abs(wildcard[myID]);
        // If this is alive then mark it
        if (myEntry > DEME_TINY_FLOAT) {
            sentry[myID] = 1;
        } else {
            sentry[myID] = 0;
        }
    }
}
