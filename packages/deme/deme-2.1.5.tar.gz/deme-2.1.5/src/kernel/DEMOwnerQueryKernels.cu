#include "hip/hip_runtime.h"
// DEM kernels used for quarrying (statistical) information from the current simulation system
#include <DEM/Defines.h>
#include <DEMHelperKernels.cuh>
_kernelIncludes_;

// Mass properties are below, if jitified mass properties are in use
_massDefs_;
_moiDefs_;
_volumeDefs_;

__global__ void inspectOwnerProperty(deme::DEMDataDT* granData,
                                     deme::DEMSimParams* simParams,
                                     float* quantity,
                                     deme::notStupidBool_t* not_in_region,
                                     size_t nOwnerBodies,
                                     deme::ownerType_t owner_type) {
    deme::bodyID_t myOwner = blockIdx.x * blockDim.x + threadIdx.x;
    if (myOwner < nOwnerBodies) {
        deme::ownerType_t myType = granData->ownerTypes[myOwner];
        if (myType & owner_type) {
            float oriQw, oriQx, oriQy, oriQz;
            double ownerX, ownerY, ownerZ;
            float myMass;
            float3 myMOI;
            // Get my mass info from either jitified arrays or global memory
            // Outputs myMass
            // Use an input named exactly `myOwner' which is the id of this owner
            { _massAcqStrat_; }

            // Get my mass info from either jitified arrays or global memory
            // Outputs myMOI
            // Use an input named exactly `myOwner' which is the id of this owner
            { _moiAcqStrat_; }

            voxelIDToPosition<double, deme::voxelID_t, deme::subVoxelPos_t>(
                ownerX, ownerY, ownerZ, granData->voxelID[myOwner], granData->locX[myOwner], granData->locY[myOwner],
                granData->locZ[myOwner], _nvXp2_, _nvYp2_, _voxelSize_, _l_);
            oriQw = granData->oriQw[myOwner];
            oriQx = granData->oriQx[myOwner];
            oriQy = granData->oriQy[myOwner];
            oriQz = granData->oriQz[myOwner];

            // Use sphereXYZ to determine if this sphere is in the region that should be counted
            // And don't forget adding LBF as an offset
            float X = ownerX + simParams->LBFX;
            float Y = ownerY + simParams->LBFY;
            float Z = ownerZ + simParams->LBFZ;
            { _inRegionPolicy_; }

            // Now it's a problem of what quantity to query
            { _quantityQueryProcess_; }
        } else {
            not_in_region[myOwner] = 1;
        }
    }
}
